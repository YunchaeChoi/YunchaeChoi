#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void add(int *a, int *b, int *c, int n)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n)
        c[index] = a[index] + b[index];
}


void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
       a[i] = rand();
}

#define N (2048*2048)
#define THREADS_PER_BLOCK 512

int main(void)
{
    int *a,*b,*c;
    int *d_a, *d_b, *d_c;
    int size = N*sizeof(int);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);

    a=(int *)malloc(size); random_ints(a,N);
    b=(int *)malloc(size); random_ints(b,N);
    c=(int *)malloc(size);

    *a=2;
    *b=5;
    printf("%d %d\n",*a,*b);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    add<<<(N +M-1)/M, M>>>(d_a,d_b,d_c,N);

    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    printf("c : %d\n",*c);

    free(a); free(b); free(c);

    hipFree(d_a); 
    hipFree(d_b); 
    hipFree(d_c); 

    return 0;
}
