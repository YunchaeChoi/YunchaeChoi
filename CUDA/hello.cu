#include <stdio.h>
#include <hip/hip_runtime.h>

#include <unistd.h>

__global__ void hello()
{
    printf("hello world from GPU\n");
}

int main()
{
    hello<<<1,4>>>();
    hipDeviceSynchronize();
    hipDeviceReset();
    printf("CPU\n");

    return 0;
}
